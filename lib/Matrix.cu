#include "Matrix.cuh"
#include "CudaFunctions.cuh"

namespace NN {
    void Matrix::initRand() const {
        GetRand(*this);
    }

    void Matrix::resize(int w, int h) {
        width = w;
        height = h;
        elements = new float[w * h];
        hipMalloc(&d_elements, sizeof(float) * w * h);
    }

    void Matrix::cpDtoH() const {
        hipMemcpy(elements, d_elements, sizeof(float) * width * height, hipMemcpyDeviceToHost);
    }

    void Matrix::cpHoD() const {
        hipMemcpy(d_elements, elements, sizeof(float) * width * height, hipMemcpyHostToDevice);
    }

    void Matrix::free() const {
        delete[] elements;
        hipFree(d_elements);
    }
}
