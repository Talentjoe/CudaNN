#include "Vector.cuh"

#include <iostream>
#include "CudaFunctions.cuh"

namespace NN {
    void Vector::initRandom() {
        GetRand(*this);
    }

    void Vector::resize(int s) {
        size = s;
        elements = new float[s];
        hipMalloc(&d_elements, sizeof(float) * s);
    }

    void Vector::cpDtoH() const {
        hipMemcpy(elements, d_elements, sizeof(float) * size, hipMemcpyDeviceToHost);
    }

    void Vector::cpHoD() const {
        hipMemcpy(d_elements, elements, sizeof(float) * size, hipMemcpyHostToDevice);
    }

    void Vector::free() const {
        delete[] elements;
        hipFree(d_elements);
    }

    void Vector::printVec() const {
        for (int i = 0; i < size; i++) {
            std::cout << elements[i] << " ";
        }
        std::cout << std::endl;
    }
}
